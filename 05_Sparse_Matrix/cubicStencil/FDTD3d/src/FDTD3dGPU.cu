#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <algorithm>
#include <iostream>

#include "FDTD3d.h"
#include "FDTD3dGPU.h"
#include "FDTD3dGPUKernel.cuh"

#define GPU_PROFILING

bool getTargetDeviceGlobalMemSize(memsize_t *result, const int argc,
                                  const char **argv) {
  int deviceCount = 0;
  int targetDevice = 0;
  size_t memsize = 0;

  // Get the number of CUDA enabled GPU devices
  printf(" hipGetDeviceCount\n");
  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  // Select target device (device 0 by default)
  targetDevice = findCudaDevice(argc, (const char **)argv);

  // Query target device for maximum memory allocation
  printf(" hipGetDeviceProperties\n");
  struct hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));

  memsize = deviceProp.totalGlobalMem;

  // Save the result
  *result = (memsize_t)memsize;
  return true;
}

bool fdtdGPU(float *output, const float *input, const float *coeff,
             const int dimx, const int dimy, const int dimz, const int radius,
             const int timesteps, const int argc, const char **argv) {
  const int outerDimx = dimx + 2 * radius;
  const int outerDimy = dimy + 2 * radius;
  const int outerDimz = dimz + 2 * radius;
  const size_t volumeSize = outerDimx * outerDimy * outerDimz;
  int deviceCount = 0;
  int targetDevice = 0;
  float *bufferOut = 0;
  float *bufferIn = 0;
  dim3 dimBlock;
  dim3 dimGrid;

  // Ensure that the inner data starts on a 128B boundary
  const int padding = (128 / sizeof(float)) - radius;
  const size_t paddedVolumeSize = volumeSize + padding;

#ifdef GPU_PROFILING
  hipEvent_t profileStart = 0;
  hipEvent_t profileEnd = 0;
  const int profileTimesteps = timesteps - 1;

  if (profileTimesteps < 1) {
    printf(
        " cannot profile with fewer than two timesteps (timesteps=%d), "
        "profiling is disabled.\n",
        timesteps);
  }

#endif

  // Check the radius is valid
  if (k_radius_min > radius && radius > k_radius_max) {
    printf("radius is invalid, must between %d and %d \n", k_radius_min,
           k_radius_max);
    exit(EXIT_FAILURE);
  }

  // Get the number of CUDA enabled GPU devices
  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  // Select target device (device 0 by default)
  targetDevice = findCudaDevice(argc, (const char **)argv);

  checkCudaErrors(hipSetDevice(targetDevice));

  // Allocate memory buffers
  checkCudaErrors(
      hipMalloc((void **)&bufferOut, paddedVolumeSize * sizeof(float)));
  checkCudaErrors(
      hipMalloc((void **)&bufferIn, paddedVolumeSize * sizeof(float)));

  // Check for a command-line specified block size
  int userBlockSize;

  if (checkCmdLineFlag(argc, (const char **)argv, "block-size")) {
    userBlockSize = getCmdLineArgumentInt(argc, argv, "block-size");
    // Constrain to a multiple of k_blockDimX
    userBlockSize = (userBlockSize / k_blockDimX * k_blockDimX);

    // Constrain within allowed bounds
    userBlockSize = MIN(MAX(userBlockSize, k_blockSizeMin), k_blockSizeMax);
  } else {
    userBlockSize = k_blockSizeMax;
  }

  // Check the device limit on the number of threads
  struct hipFuncAttributes funcAttrib;
  checkCudaErrors(
      hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(FiniteDifferencesKernel<4>)));

  userBlockSize = MIN(userBlockSize, funcAttrib.maxThreadsPerBlock);

  // Set the block size
  dimBlock.x = k_blockDimX;
  // Visual Studio 2005 does not like std::min
  //    dimBlock.y = std::min<size_t>(userBlockSize / k_blockDimX,
  //    (size_t)k_blockDimMaxY);
  dimBlock.y = ((userBlockSize / k_blockDimX) < (size_t)k_blockDimMaxY)
                   ? (userBlockSize / k_blockDimX)
                   : (size_t)k_blockDimMaxY;
  dimGrid.x = (unsigned int)ceil((float)dimx / dimBlock.x);
  dimGrid.y = (unsigned int)ceil((float)dimy / dimBlock.y);
  printf(" set block size to %dx%d\n", dimBlock.x, dimBlock.y);
  printf(" set grid size to %dx%d\n", dimGrid.x, dimGrid.y);

  // Check the block size is valid
  if (dimBlock.x < radius || dimBlock.y < radius) {
    printf("invalid block size, x (%d) and y (%d) must be >= radius (%d).\n",
           dimBlock.x, dimBlock.y, radius);
    exit(EXIT_FAILURE);
  }

  // Copy the input to the device input buffer
  checkCudaErrors(hipMemcpy(bufferIn + padding, input,
                             volumeSize * sizeof(float),
                             hipMemcpyHostToDevice));

  // Copy the input to the device output buffer (actually only need the halo)
  checkCudaErrors(hipMemcpy(bufferOut + padding, input,
                             volumeSize * sizeof(float),
                             hipMemcpyHostToDevice));

  // Copy the coefficients to the device coefficient buffer
  bool use_kernel2 = false;
  if (checkCmdLineFlag(argc, (const char **)argv, "kernel2")) {
    use_kernel2 = true;
  }

  if (!use_kernel2) {
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(stencil), (void *)coeff,
                                       (radius + 1) * sizeof(float)));
  } else {
    int sten_dim = 2 * k_radius_max + 1;
    int sten_dim_len = sten_dim * sten_dim * sten_dim;
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(stencil2), (void *)coeff,
                                       sten_dim_len * sizeof(float)));
  }

#ifdef GPU_PROFILING

  // Create the events
  checkCudaErrors(hipEventCreate(&profileStart));
  checkCudaErrors(hipEventCreate(&profileEnd));

#endif

  // Execute the FDTD
  float *bufferSrc = bufferIn + padding;
  float *bufferDst = bufferOut + padding;
  printf(" GPU FDTD loop\n");

#ifdef GPU_PROFILING
  // Enqueue start event
  checkCudaErrors(hipEventRecord(profileStart, 0));
#endif
  int shMeme_size = (dimBlock.x + 2 * radius) * (dimBlock.y + 2 * radius);
  shMeme_size *= sizeof(float);
  for (int it = 0; it < timesteps; it++) {
    printf("\tt = %d ", it);

    // Launch the kernel
    if (use_kernel2) {
      printf("launch kernel2\n");
      switch (radius) {
        case 1:
          FiniteDifferencesKernel2<1><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 2:
          FiniteDifferencesKernel2<2><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 3:
          FiniteDifferencesKernel2<3><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 4:
          FiniteDifferencesKernel2<4><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 5:
          FiniteDifferencesKernel2<5><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 6:
          FiniteDifferencesKernel2<6><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 7:
          FiniteDifferencesKernel2<7><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 8:
          FiniteDifferencesKernel2<8><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 9:
          FiniteDifferencesKernel2<9><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 10:
          FiniteDifferencesKernel2<10><<<dimGrid, dimBlock, shMeme_size>>>(
              bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        default:
          break;
      }
    } else {
      printf("launch kernel1\n");
      switch (radius) {
        case 1:
          FiniteDifferencesKernel<1>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 2:
          FiniteDifferencesKernel<2>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 3:
          FiniteDifferencesKernel<3>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 4:
          FiniteDifferencesKernel<4>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 5:
          FiniteDifferencesKernel<5>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 6:
          FiniteDifferencesKernel<6>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 7:
          FiniteDifferencesKernel<7>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 8:
          FiniteDifferencesKernel<8>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 9:
          FiniteDifferencesKernel<9>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        case 10:
          FiniteDifferencesKernel<10>
              <<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
          break;
        default:
          break;
      }
    }

    // Toggle the buffers
    // Visual Studio 2005 does not like std::swap
    //    std::swap<float *>(bufferSrc, bufferDst);
    float *tmp = bufferDst;
    bufferDst = bufferSrc;
    bufferSrc = tmp;
  }

  printf("\n");

#ifdef GPU_PROFILING
  // Enqueue end event
  checkCudaErrors(hipEventRecord(profileEnd, 0));
#endif

  // Wait for the kernel to complete
  checkCudaErrors(hipDeviceSynchronize());

  // Read the result back, result is in bufferSrc (after final toggle)
  checkCudaErrors(hipMemcpy(output, bufferSrc, volumeSize * sizeof(float),
                             hipMemcpyDeviceToHost));

// Report time
#ifdef GPU_PROFILING
  float elapsedTimeMS = 0;

  if (profileTimesteps > 0) {
    checkCudaErrors(
        hipEventElapsedTime(&elapsedTimeMS, profileStart, profileEnd));
  }

  if (profileTimesteps > 0) {
    // Convert milliseconds to seconds
    double elapsedTime = elapsedTimeMS * 1.0e-3;
    double avgElapsedTime = elapsedTime / (double)profileTimesteps;
    // Determine number of computations per timestep
    size_t pointsComputed = dimx * dimy * dimz;
    // Determine throughput
    double throughputM = 1.0e-6 * (double)pointsComputed / avgElapsedTime;
    printf(
        "FDTD3d, Throughput = %.4f MPoints/s, Time = %.5f s, Size = %lu "
        "Points, "
        "NumDevsUsed = %u, Blocksize = %u\n",
        throughputM, avgElapsedTime, pointsComputed, 1,
        dimBlock.x * dimBlock.y);
  }

#endif

  // Cleanup
  if (bufferIn) {
    checkCudaErrors(hipFree(bufferIn));
  }

  if (bufferOut) {
    checkCudaErrors(hipFree(bufferOut));
  }

#ifdef GPU_PROFILING

  if (profileStart) {
    checkCudaErrors(hipEventDestroy(profileStart));
  }

  if (profileEnd) {
    checkCudaErrors(hipEventDestroy(profileEnd));
  }

#endif
  return true;
}

bool fdtdGPU_BandedMat(float *output, const float *input,
             const float **coeff_buffers,
             const int dimx, const int dimy, const int dimz, const int radius,
             const int timesteps, const int argc, const char **argv) {
  const int outerDimx = dimx + 2 * radius;
  const int outerDimy = dimy + 2 * radius;
  const int outerDimz = dimz + 2 * radius;
  const size_t volumeSize = outerDimx * outerDimy * outerDimz;
  int deviceCount = 0;
  int targetDevice = 0;
  float *bufferOut = 0;
  float *bufferIn = 0;
  dim3 dimBlock;
  dim3 dimGrid;

  // Ensure that the inner data starts on a 128B boundary
  const int padding = (128 / sizeof(float)) - radius;
  const size_t paddedVolumeSize = volumeSize + padding;

#ifdef GPU_PROFILING
  hipEvent_t profileStart = 0;
  hipEvent_t profileEnd = 0;
  const int profileTimesteps = timesteps - 1;

  if (profileTimesteps < 1) {
    printf(
        " cannot profile with fewer than two timesteps (timesteps=%d), "
        "profiling is disabled.\n",
        timesteps);
  }

#endif

  // Check the radius is valid
  if (k_radius_min > radius && radius > k_radius_max) {
    printf("radius is invalid, must between %d and %d \n", k_radius_min,
           k_radius_max);
    exit(EXIT_FAILURE);
  }

  // Get the number of CUDA enabled GPU devices
  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  // Select target device (device 0 by default)
  targetDevice = findCudaDevice(argc, (const char **)argv);

  checkCudaErrors(hipSetDevice(targetDevice));

  // Allocate memory buffers
  checkCudaErrors(
      hipMalloc((void **)&bufferOut, paddedVolumeSize * sizeof(float)));
  checkCudaErrors(
      hipMalloc((void **)&bufferIn, paddedVolumeSize * sizeof(float)));

  // Check for a command-line specified block size
  int userBlockSize;

  if (checkCmdLineFlag(argc, (const char **)argv, "block-size")) {
    userBlockSize = getCmdLineArgumentInt(argc, argv, "block-size");
    // Constrain to a multiple of k_blockDimX
    userBlockSize = (userBlockSize / k_blockDimX * k_blockDimX);

    // Constrain within allowed bounds
    userBlockSize = MIN(MAX(userBlockSize, k_blockSizeMin), k_blockSizeMax);
  } else {
    userBlockSize = k_blockSizeMax;
  }

  // Check the device limit on the number of threads
  struct hipFuncAttributes funcAttrib;
  checkCudaErrors(
      hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(FiniteDifferencesKernel<4>)));

  userBlockSize = MIN(userBlockSize, funcAttrib.maxThreadsPerBlock);

  // Set the block size
  dimBlock.x = k_blockDimX;
  // Visual Studio 2005 does not like std::min
  //    dimBlock.y = std::min<size_t>(userBlockSize / k_blockDimX,
  //    (size_t)k_blockDimMaxY);
  dimBlock.y = ((userBlockSize / k_blockDimX) < (size_t)k_blockDimMaxY)
                   ? (userBlockSize / k_blockDimX)
                   : (size_t)k_blockDimMaxY;
  dimGrid.x = (unsigned int)ceil((float)dimx / dimBlock.x);
  dimGrid.y = (unsigned int)ceil((float)dimy / dimBlock.y);
  printf(" set block size to %dx%d\n", dimBlock.x, dimBlock.y);
  printf(" set grid size to %dx%d\n", dimGrid.x, dimGrid.y);

  // Check the block size is valid
  if (dimBlock.x < radius || dimBlock.y < radius) {
    printf("invalid block size, x (%d) and y (%d) must be >= radius (%d).\n",
           dimBlock.x, dimBlock.y, radius);
    exit(EXIT_FAILURE);
  }

  // Allocate coeff buffers on GPU
  int num_coeff = 2*radius + 1;
  num_coeff = pow(num_coeff, 3);
  float **d_buffers = 0;
  checkCudaErrors(
      hipMalloc((void **)&d_buffers, num_coeff * sizeof(float*)));

  // temporary store the device addresses of the buffers
  float **hd_buffers;
  hd_buffers = (float**)calloc(num_coeff, sizeof(float*));

  size_t innerVolumeSize = dimx * dimy * dimz;
  for (int i = 0; i < num_coeff; ++i) {
      checkCudaErrors(
        hipMalloc((void **)&(hd_buffers[i]), innerVolumeSize * sizeof(float)));
  }
  
  // Copy the coeff buffer pointers to the device
  checkCudaErrors(hipMemcpy(d_buffers, hd_buffers,
                             num_coeff * sizeof(float*),
                             hipMemcpyHostToDevice));

  for (int i = 0; i < num_coeff; ++i) {
    // Copy the actual coeff buffers to the device
    checkCudaErrors(hipMemcpy(hd_buffers[i], coeff_buffers[i],
                              innerVolumeSize * sizeof(float),
                              hipMemcpyHostToDevice));
  }

  // Copy the input to the device input buffer
  checkCudaErrors(hipMemcpy(bufferIn + padding, input,
                             volumeSize * sizeof(float),
                             hipMemcpyHostToDevice));

  // Copy the input to the device output buffer (actually only need the halo)
  checkCudaErrors(hipMemcpy(bufferOut + padding, input,
                             volumeSize * sizeof(float),
                             hipMemcpyHostToDevice));


#ifdef GPU_PROFILING

  // Create the events
  checkCudaErrors(hipEventCreate(&profileStart));
  checkCudaErrors(hipEventCreate(&profileEnd));

#endif

  // Execute the FDTD
  float *bufferSrc = bufferIn + padding;
  float *bufferDst = bufferOut + padding;
  printf(" GPU FDTD loop\n");

#ifdef GPU_PROFILING
  // Enqueue start event
  checkCudaErrors(hipEventRecord(profileStart, 0));
#endif
  int shMeme_size = (dimBlock.x + 2 * radius) * (dimBlock.y + 2 * radius);
  shMeme_size *= sizeof(float);
  for (int it = 0; it < timesteps; it++) {
    printf("\tt = %d ", it);

    // Launch the kernel
    printf("launch kernel2\n");
    switch (radius) {
      case 1:
        FiniteDifferencesKernel3<1><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      case 2:
        FiniteDifferencesKernel3<2><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      case 3:
        FiniteDifferencesKernel3<3><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      case 4:
        FiniteDifferencesKernel3<4><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      case 5:
        FiniteDifferencesKernel3<5><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      case 6:
        FiniteDifferencesKernel3<6><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      case 7:
        FiniteDifferencesKernel3<7><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      case 8:
        FiniteDifferencesKernel3<8><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      case 9:
        FiniteDifferencesKernel3<9><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      case 10:
        FiniteDifferencesKernel3<10><<<dimGrid, dimBlock, shMeme_size>>>(
            bufferDst, bufferSrc, dimx, dimy, dimz, d_buffers);
        break;
      default:
        break;
    }

    // Toggle the buffers
    // Visual Studio 2005 does not like std::swap
    //    std::swap<float *>(bufferSrc, bufferDst);
    float *tmp = bufferDst;
    bufferDst = bufferSrc;
    bufferSrc = tmp;
  }

  printf("\n");

#ifdef GPU_PROFILING
  // Enqueue end event
  checkCudaErrors(hipEventRecord(profileEnd, 0));
#endif

  // Wait for the kernel to complete
  checkCudaErrors(hipDeviceSynchronize());

  // Read the result back, result is in bufferSrc (after final toggle)
  checkCudaErrors(hipMemcpy(output, bufferSrc, volumeSize * sizeof(float),
                             hipMemcpyDeviceToHost));

// Report time
#ifdef GPU_PROFILING
  float elapsedTimeMS = 0;

  if (profileTimesteps > 0) {
    checkCudaErrors(
        hipEventElapsedTime(&elapsedTimeMS, profileStart, profileEnd));
  }

  if (profileTimesteps > 0) {
    // Convert milliseconds to seconds
    double elapsedTime = elapsedTimeMS * 1.0e-3;
    double avgElapsedTime = elapsedTime / (double)profileTimesteps;
    // Determine number of computations per timestep
    size_t pointsComputed = dimx * dimy * dimz;
    // Determine throughput
    double throughputM = 1.0e-6 * (double)pointsComputed / avgElapsedTime;
    printf(
        "FDTD3d, Throughput = %.4f MPoints/s, Time = %.5f s, Size = %lu "
        "Points, "
        "NumDevsUsed = %u, Blocksize = %u\n",
        throughputM, avgElapsedTime, pointsComputed, 1,
        dimBlock.x * dimBlock.y);
  }

#endif

  // Cleanup
  if (bufferIn) {
    checkCudaErrors(hipFree(bufferIn));
  }

  if (bufferOut) {
    checkCudaErrors(hipFree(bufferOut));
  }

  if (d_buffers){
    for (int i = 0; i < num_coeff; ++i) {
      if(hd_buffers[i]){
        checkCudaErrors(hipFree(hd_buffers[i]));
      }
    }
    checkCudaErrors(hipFree(d_buffers));
  }


#ifdef GPU_PROFILING

  if (profileStart) {
    checkCudaErrors(hipEventDestroy(profileStart));
  }

  if (profileEnd) {
    checkCudaErrors(hipEventDestroy(profileEnd));
  }

#endif
  return true;
}